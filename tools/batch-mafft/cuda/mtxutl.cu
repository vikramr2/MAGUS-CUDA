
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stddef.h>

void FreeCharMtx( char **mtx )
{
	int i;
	for( i=0; mtx[i]; i++ ) 
	{
		hipFree( mtx[i] );
	}
	hipFree( mtx );
}

char *AllocateCharVec( int l1 )
{
	char *cvec;
	
    hipMalloc((void**) &cvec, l1*sizeof(char));
	if (!cvec) exit(1);
	return( cvec );
}

char **AllocateCharMtx( int l1, int l2 )
{
	int i;
	char **cmtx;
	
    hipMalloc((void**) cmtx, (l1+1)*sizeof(char*));
	if( cmtx == NULL ) exit(1);

	if( l2 )
	{
		for( i=0; i<l1; i++ ) 
		{
			cmtx[i] = AllocateCharVec( l2 );
		}
	}
	cmtx[l1] = NULL;
	return( cmtx );
} 

int *AllocateIntVec( int ll1 )
{
	int *vec;

	hipMalloc((void**) &vec, ll1*sizeof(int));
	if( vec == NULL ) exit(1);
	return( vec );
}	