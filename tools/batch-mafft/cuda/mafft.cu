#include "replaceu.c"

#define er 0
#define progname "linsi"
#define anysymbol 1
#define ep 0.123

// linsi properties
#define fft 0
#define cycle 1
#define iterate 1000
#define distance "local"
#define strategy "L-INS-i"

// other properties to keep track of
#define addfile NULL
#define tmpfile "/var/folders/xb/ttq_ycbj3x7013f1p015bhv00000gn/T/mafft-linsi.XXXXXXXXXX.OqQAktC0"
#define seed NULL
#define seednseq 0
#define numthreads 0
#define numthreadstb 0
#define numthreadsit 0
#define nadd 0
#define gopdist 1.53
#define aof -0.123
#define laof 0.1
#define pgaof 0.1
#define iteratelimit 16
//#define iterate 16
#define rnaopt "  "
#define rnaoptit " -F "
#define swopt " "
#define parttreeoutopt " "
#define treeoutopt " "
#define distoutopt " "
#define formatcheck 0
#define fragment 0
#define pairspecified 0
#define localparam "-l 2.7"
#define param_fft " "
#define cycletbfast 1
#define cycledisttbfast 1
#define outputformat "pir"
#define outputopt "-f -l 60"

static int unaligned_charlen = 0;

// TODO: ​mafft-linsi --ep 0.123 --quiet --anysymbol input > output
__host__
char* extract_file_contents(char* filename) {
    FILE *f;
    char s;

    f=fopen(filename,"r");
    int len = 0;
    while((s=fgetc(f))!=EOF) {
        len++;
    }

    char* ret = malloc((len+1)*sizeof(char));
    int i = 0;
    rewind(f);
    while ((s=fgetc(f))!=EOF) {
        ret[i] = s;
        i++;
    }

    unaligned_charlen = i;

    fclose(f);
    return ret;
}

__device__
char* mafft_magus(char* unaligned, int nseq) {
    return "pass";
}

int main(int argc, char* argv[]) {
    char* inputfile = "../mafft-main/test/sample";
    char* unaligned = extract_file_contents(inputfile);
    char dorp = 'n';
    int njob = 0;
    int nlenmax = 0;
    int nlenmin = 0;

    char** seq;
    char** name;
    int* nlen;

    char* d_unaligned; 
    char* d_dorp;
    int* d_njob;
    int* d_nlenmax;
    int* d_nlenmin;

    hipMalloc((void**) &d_unaligned, unaligned_charlen*sizeof(char));
    hipMalloc((void**) &d_dorp, sizeof(char));
    hipMalloc((void**) &d_njob, sizeof(int));
    hipMalloc((void**) &d_nlenmax, sizeof(int));
    hipMalloc((void**) &d_nlenmin, sizeof(int));

    hipMemcpy(d_unaligned, unaligned, unaligned_charlen*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_dorp, &dorp, sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_njob, &njob, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_nlenmax, &nlenmax, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_nlenmin, &nlenmin, sizeof(int), hipMemcpyHostToDevice);

    replaceu<<<1, 1>>>(d_unaligned, unaligned_charlen, d_dorp, d_njob, d_nlenmax, d_nlenmin);

    hipMemcpy(&njob, d_njob, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&nlenmax, d_nlenmax, sizeof(int), hipMemcpyDeviceToHost);

    seq = AllocateCharMtx( *njob, (*nlenmax)+1 );
	name = AllocateCharMtx( *njob, B+1 );
	nlen = AllocateIntVec( *njob );

    hipMemcpy(d_dorp, &dorp, sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_njob, &njob, sizeof(int), hipMemcpyHostToDevice);

    replaceu2<<<1, 1>>>(d_unaligned, unaligned_charlen, d_dorp, d_njob, d_nlenmax, d_nlenmin, seq, name, nlen);

    hipMemcpy(unaligned, d_unaligned, unaligned_charlen*sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(&dorp, d_dorp, sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(&njob, d_njob, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&nlenmax, d_nlenmax, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&nlenmin, d_nlenmin, sizeof(int), hipMemcpyDeviceToHost);

    fprintf(stderr, "dorp:\t%c\nnjob:\t%d\nnlenmax:\t%d\nnlenmin:\t%d\n", dorp, njob, nlenmax, nlenmin);
}